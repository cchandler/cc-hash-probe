#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gpu.h"

/*

High performance credit card hash probe using GPU.  Borrowed segments of Steve Worley's
SHA1 function from the EngineYard contest.

Thanks:
Steve Worley < m a t h g e e k@(my last name).com >
*/



#define SIZE     10000128

unsigned long long int *d_intervals;
	
unsigned int *d_start_positions_msd;
unsigned int *d_start_positions_lsd;

unsigned int *d_num1;
unsigned int *d_num2;

int* h_valid;
unsigned int* d_valid;
unsigned int* d_hash;

// __host__ __device__ unsigned int swapends(unsigned int v) 
// {
//   return 
//     ((255&(v>> 0))<<24)+
//     ((255&(v>> 8))<<16)+
//     ((255&(v>>16))<<8)+
//     ((255&(v>>24))<<0);
// }

/* 
   From Steve's notes:
   We don't want to precompute and store all 80 w array
   values. Instead we store only the next 16 values and update them in
   a logrolling array. Complicated but it means we can fit the tables
   in shared memory */
__device__ unsigned int popNextW(unsigned int *w, int &wIndex)
{
  unsigned int nextW=w[wIndex&15];
  int thisIndex=wIndex&15;
  w[thisIndex]^=w[(wIndex+16-3)&15]^w[(wIndex+16-8)&15]^w[(wIndex+16-14)&15];
  w[thisIndex]=  (w[thisIndex]<<1) | (w[thisIndex]>>31);
  ++wIndex;

  //  if (threadIdx.x==0) debugprint("pop %08x\n", nextW);
  return nextW;
}

__device__ unsigned int popFinalWs(unsigned int *w, int &wIndex)
{
  unsigned int nextW=w[wIndex&15];
  ++wIndex;
  return nextW;
}

__device__ int generateHash(unsigned int num1, unsigned int num2, unsigned int *hash){
	extern __shared__ unsigned int fullw[];
	
	int hash_offset = threadIdx.x + blockIdx.x * blockDim.x;
	
	unsigned int d_initVector[5];
	d_initVector[0] = 0x67452301;
	d_initVector[1] = 0xEFCDAB89;
	d_initVector[2] = 0x98BADCFE;
	d_initVector[3] = 0x10325476;
	d_initVector[4] = 0xC3D2E1F0;
	
	// unsigned int *w=fullw+17*threadIdx.x; // spaced by 17 to avoid bank conflicts, CC: TODO verify this
	char lookup_table[10] = {48,49,50,51,52,53,54,55,56,57};
	int pos = 0;
	unsigned int digit = 0;
	unsigned int num_1a = 0;
	unsigned int num_2a = 0;
	unsigned int num_3a = 0;
	unsigned int num_4a = 0;
	
	#pragma unroll 999
	for(pos = 0; pos <= 3; ++pos) {
		digit = 0;
		digit = num2 & (0xF << (pos * 4));
		digit = digit >> (pos * 4);
		
		num_1a = num_1a | lookup_table[digit];
		if(pos != 3) {num_1a = num_1a << 8;};
	}
	
	#pragma unroll 999
	for(pos = 4; pos <= 7; ++pos) {
		digit = 0;
		digit = num2 & (0xF << (pos * 4));
		digit = digit >> (pos * 4);
		
		num_2a = num_2a | lookup_table[digit];
		if(pos != 7) {num_2a = num_2a << 8;};
	}
	
	#pragma unroll 999
	for(pos = 0; pos <= 3; ++pos) {
		digit = 0;
		digit = num1 & (0xF << (pos * 4));
		digit = digit >> (pos * 4);
		
		num_3a = num_3a | lookup_table[digit];
		if(pos != 3) {num_3a = num_3a << 8;};
	}
	
	#pragma unroll 999
	for(pos = 4; pos <= 7; ++pos) {
		digit = 0;
		digit = num1 & (0xF << (pos * 4));
		digit = digit >> (pos * 4);
		
		num_4a = num_4a | lookup_table[digit];
		if(pos != 7) {num_4a = num_4a << 8;};
	}
	
	
	unsigned int w[80] = {'\0'};
	for (int i=0; i<80; i++) { w[i] = '\0'; };
	// w[0] = 1633837952; // 'abc' + 1 bit
	// num_1a = num_1a << 8;
	w[0] = num_1a;
	w[1] = num_2a;
	w[2] = num_3a;
	w[3] = num_4a;
	w[4] = (unsigned) 8 << 28;
	w[15] = 128;
	
	int wIndex=0;
	
	
	unsigned int a = d_initVector[0];
	unsigned int b = d_initVector[1];
    unsigned int c = d_initVector[2];
    unsigned int d = d_initVector[3];
    unsigned int e = d_initVector[4];
	
	#pragma unroll 999
	for (int i=0; i<20; ++i) {
	  unsigned int thisW=popNextW(w, wIndex);
	  // unsigned int thisW=w[i];
	  //    unsigned int f= (b&c)|((~b)&d);
	  unsigned int f= d ^ (b & (c^d)); // alternate computation of above
	  unsigned int temp=((a<<5)|(a>>27))+f+e+0x5A827999+thisW;
	  e=d;
	  d=c;
	  c=(b<<30)|(b>>2);
	  b=a;
	  a=temp;
	}
	
	#pragma unroll 999
	for (int i=20; i<40; ++i) {
	  unsigned int thisW=popNextW(w, wIndex);
	  // unsigned int thisW=w[i];
	  unsigned int f= b^c^d;
	  unsigned int temp=((a<<5)|(a>>27))+f+e+0x6ED9EBA1+thisW;
	  e=d;
	  d=c;
	  c=(b<<30)|(b>>2);
	  b=a;
	  a=temp;
	}

	#pragma unroll 999
	for (int i=40; i<60; ++i) {
	  unsigned int thisW=popNextW(w, wIndex);
	  // unsigned int thisW=w[i];
	  //    unsigned int f= (b&c) | (b&d) | (c&d);
	  unsigned int f= (b&c) | (d & (b|c)); // alternate computation of above
	  unsigned int temp=((a<<5)|(a>>27))+f+e+0x8F1BBCDC+thisW;
	  e=d;
	  d=c;
	  c=(b<<30)|(b>>2);
	  b=a;
	  a=temp;
	}

	#pragma unroll 999
	for (int i=60; i<64; ++i) {
	  unsigned int thisW=popNextW(w, wIndex);
	// unsigned int thisW=w[i];
	  unsigned int f= b^c^d;
	  unsigned int temp=((a<<5)|(a>>27))+f+e+0xCA62C1D6+thisW;
	  e=d;
	  d=c;
	  c=(b<<30)|(b>>2);
	  b=a;
	  a=temp;
	}


	#pragma unroll 999
	for (int i=64; i<80; ++i) {
	  unsigned int thisW=popFinalWs(w, wIndex); // simpler compute for final rounds
	  // unsigned int thisW=w[i];
	  unsigned int f= b^c^d;
	  unsigned int temp=((a<<5)|(a>>27))+f+e+0xCA62C1D6+thisW;
	  e=d;
	  d=c;
	  c=(b<<30)|(b>>2);
	  b=a;
	  a=temp;
	}
	
	hash[hash_offset*5 + 0] = a + d_initVector[0];
	hash[hash_offset*5 + 1] = b + d_initVector[1];
	hash[hash_offset*5 + 2] = c + d_initVector[2];
	hash[hash_offset*5 + 3] = d + d_initVector[3];
	hash[hash_offset*5 + 4] = e + d_initVector[4];
	
	return 0;
}

__device__ unsigned long long int GPUbitPackCC(unsigned long long int num){
	int i = 0;
	int digit = 0;
	unsigned long long int result = 0;
	for(i = 15; i >= 0; --i){
		digit = num % 10;
		num = num / 10;
		result = result << 4;
		result = result | digit;
	}
	return result;
}

__device__ void GPULuhn(unsigned int *num1, unsigned int *num2,unsigned int *valid, size_t valid_pitch){
	int pos = 0;
	unsigned int digit = 0;
	int even = 0;
	unsigned int sum = 0;
	int lookup_table[10] = {0,2,4,6,8,1,3,5,7,9};
	
	for(pos = 7; pos >= 0; --pos) {
		digit = 0;
		digit = *num1 & (0xF << (pos * 4));
		digit = digit >> (pos * 4);
		
		if(even) {
			digit = lookup_table[digit];
		}
		sum = sum + digit;
		even = !even;
	}
	for(pos = 7; pos >= 0; --pos) {
		digit = 0;
		digit = *num2 & (0xF << (pos * 4));
		digit = digit >> (pos * 4);
		
		if(even) {
			digit = lookup_table[digit];
		}
		sum = sum + digit;
		even = !even;
	}
	
	*valid = (sum % 10 == 0);
}

__global__ void GPUProbe(unsigned long long int *intervals, unsigned int *start_positions_msd,unsigned int *start_positions_lsd,unsigned int *num1, unsigned int *num2,unsigned int *valid, size_t valid_pitch, unsigned int *hash){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	unsigned long long int j = intervals[blockIdx.x] + threadIdx.x;
	j = GPUbitPackCC(j);
	
	unsigned int num_lsd = j;
	j = j >> 32;
	unsigned int num_msd = j;
	
	start_positions_lsd[threadIdx.x] = num_msd;
	
	unsigned int* valid_row = (unsigned int*)((char*)valid + 0 * valid_pitch); // 0 is the height offset. zero right now because it's essentially linear
	// unsigned int* num1_row = (unsigned int*)((char*)num1 + 0 * valid_pitch);
	// unsigned int* num2_row = (unsigned int*)((char*)num2 + 0 * valid_pitch);
	// 
	// num1_row[i] = num_msd;
	// num2_row[i] = num_lsd;
	
	
	// GPULuhn(&num1_row[i],&num2_row[i],&valid_row[i],valid_pitch); //TODO Remove dereference operator
	// generateHash(num1_row[i],num2_row[i],hash);
	
	GPULuhn(&num_msd,&num_lsd,&valid_row[i],valid_pitch);
	generateHash(num_msd,num_lsd,hash);
}

int setupCUDA(){
	int error = 0;
	int deviceCount = 0;
	error = hipGetDeviceCount(&deviceCount);
	if (error != hipSuccess) {
		printf("The system is reporting no devices available... %d\n",error);
		exit(-1);
	}
	
	error = hipSetDevice(0);
	if(error != hipSuccess){
		printf("Unable to set runtime device... %d\n",error);
		exit(-1);
	}
	
	return 0;
}

// extern "C"{
	int test(unsigned long *intervals, unsigned int *num1, unsigned int *num2,unsigned int *h_valid)
	{	
		int error = 0;
		
		// size_t d_start_positions_msd_pitch;
		// error = hipMallocPitch((void **)(&d_start_positions_msd), &d_start_positions_msd_pitch, 192 * sizeof(int), 1); //This SIZE is the thread count
		// if(error != hipSuccess){
		// 	printf("One of the mallocPitchs failed %d \n", error);
		// 	return -1;
		// }
		// size_t d_start_positions_lsd_pitch;
		// error = hipMallocPitch((void **)(&d_start_positions_lsd), &d_start_positions_lsd_pitch, 192 * sizeof(int), 1); //This SIZE is the thread count
		// if(error != hipSuccess){
		// 	printf("One of the mallocPitchs failed %d \n", error);
		// 	return -1;
		// }
		
		size_t d_intervals_pitch;
		error = hipMallocPitch((void **)(&d_intervals), &d_intervals_pitch, blocksize * sizeof(int), 1); //This SIZE is the thread count
		if(error != hipSuccess){
			printf("One of the mallocPitchs failed %d \n", error);
			return -1;
		}
		
		// size_t d_num1_pitch;
		// 		error = hipMallocPitch((void **)(&d_num1), &d_num1_pitch, SIZE * sizeof(int), 1);
		// 		if(error != hipSuccess){
		// 			printf("One of the mallocPitchs failed %d \n", error);
		// 			return -1;
		// 		}
		// 		size_t d_num2_pitch;
		// 		error = hipMallocPitch((void **)(&d_num2), &d_num2_pitch, SIZE * sizeof(int), 1);
		// 		if(error != hipSuccess){
		// 			printf("One of the mallocPitchs failed %d \n", error);
		// 			return -1;
		// 		}
		size_t d_valid_pitch;
		error = hipMallocPitch((void **)(&d_valid), &d_valid_pitch, SIZE * sizeof(int), 1);
		if(error != hipSuccess){
			printf("One of the mallocPitchs failed %d \n", error);
			return -1;
		}
		size_t d_hash_pitch;
		error = hipMallocPitch((void **)(&d_hash), &d_hash_pitch, SIZE * sizeof(int) * 5, 1);
		
		if(error != hipSuccess){
			printf("One of the mallocPitchs failed %d \n", error);
			return -1;
		}
		
		int *h_hash = (int*)malloc(SIZE * sizeof(int) * 5);
		unsigned int *h_start_positions_lsd = (unsigned int*)malloc(192 * sizeof(unsigned int));
		
	
		// error = hipMemset2D(d_num1, d_num1_pitch, 0, SIZE * sizeof(int), 1);
		// 		error = hipMemset2D(d_num2, d_num2_pitch, 0, SIZE * sizeof(int), 1);
		error = hipMemset2D(d_valid, d_valid_pitch, 0, SIZE * sizeof(int), 1);
		error = hipMemset2D(d_hash, d_hash_pitch, 0, SIZE * sizeof(int) * 5, 1);
		
		// error = hipMemset2D(d_start_positions_msd, d_start_positions_msd_pitch, 0, 192 * sizeof(int), 1); //192 is the thread count
		// 		error = hipMemset2D(d_start_positions_lsd, d_start_positions_lsd_pitch, 0, 192 * sizeof(int), 1); //192 is the thread count
		error = hipMemset2D(d_intervals, d_intervals_pitch, 0, blocksize * sizeof(int), 1);
		
		// error = hipMemcpy2D(d_num1,d_num1_pitch, num1, sizeof(int) * 512, 512 * sizeof(int), 1, hipMemcpyHostToDevice);
		// error = hipMemcpy2D(d_num2,d_num2_pitch, num2, sizeof(int) * 512, 512 * sizeof(int), 1, hipMemcpyHostToDevice);
		error = hipMemcpy2D(d_intervals,d_intervals_pitch, intervals, sizeof(int) * blocksize, blocksize * sizeof(int), 1, hipMemcpyHostToDevice);
		
		if(error != hipSuccess){
			printf("One of the mem copies or memsets failed %d\n", error);
			return -1;
		}
	
		GPUProbe<<< blocksize , threadsize >>>(d_intervals, d_start_positions_msd,d_start_positions_lsd, d_num1,d_num2,d_valid, d_valid_pitch, d_hash);
		hipDeviceSynchronize();
		
		
		// error = hipMemcpy2D(num1, sizeof(int) * 512, d_num1, d_num1_pitch, 512 * sizeof(int), 1, hipMemcpyDeviceToHost);
		// if(error != hipSuccess){
		// 	printf("Failed to copy d_num1 from device %d\n", error);
		// }
		// error = hipMemcpy2D(num2,sizeof(int) * 512, d_num2, d_num2_pitch, 512 * sizeof(int), 1, hipMemcpyDeviceToHost);
		// if(error != hipSuccess){
		// 	printf("Failed to copy d_num2 from device %d\n", error);
		// }
		
		// error = hipMemcpy2D(h_start_positions_lsd, sizeof(int) * 192, d_start_positions_lsd, d_start_positions_lsd_pitch, 192 * sizeof(int), 1, hipMemcpyDeviceToHost);
		// if(error != hipSuccess){
		// 	printf("Failed to copy d_start_positions_lsd from device %d\n", error);
		// }
		
		error = hipMemcpy2D(h_valid,sizeof(int) * 512, d_valid, d_valid_pitch, 512 * sizeof(int), 1, hipMemcpyDeviceToHost);
		if(error != hipSuccess){
			printf("Failed to copy d_valid from device %d\n",error);
		}
		
		error = hipMemcpy2D(h_hash,sizeof(int) * 512 * 5, d_hash, d_hash_pitch, 512 * sizeof(int) * 5, 1, hipMemcpyDeviceToHost);
		if(error != hipSuccess){
			printf("Failed to copy d_hash from device %d\n",error);
			return -1;
		}
		
		if(error){
			printf("OMGWTFBBQ error %d\n",error);
			return -1;
		}
		// 
		// int i =0;
		// for(i = 0; i < 512; i++){
		// 	// if(h_valid[i]){
		// 		printf("%d --- Chunk1 %08x %08x  Valid %u\n",i, num2[i],num1[i], h_valid[i]);
		// 		printf("\tHash: %08x %08x %08x %08x %08x\n", h_hash[0 + i*5],h_hash[1 + i*5],h_hash[2 + i*5],h_hash[3 + i*5],h_hash[4 + i*5]);
		// 	// }
		// }
		// 
		// for(i = 0; i < 192; i++){
		// 	printf("%lu %u\n", intervals[i], h_start_positions_lsd[i]);
		// }
		// 
		// error = hipFree(d_num1);
		// error = hipFree(d_num2);
		error = hipFree(d_valid);
		error = hipFree(d_hash);
		// error = hipFree(d_start_positions_msd);
		// error = hipFree(d_start_positions_lsd);
		error = hipFree(d_intervals);
		
		free(h_hash);
		free(h_start_positions_lsd);
		
		error = 0;
		if(error){
			printf("OMGWTFBBQ error %d\n",error);
			return -1;
		}
		
		// printf("Success...\n");
		
		return 0;
	}
// }
